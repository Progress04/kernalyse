
#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel for vector addition
__global__ void vecAdd(float *a, float *b, float *c, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;  //blockIdx = block index, blockDim = number of threads in a block, threadIdx = thread index within a block
  if (i < n) c[i] = a[i] + b[i];
}

int main() {
  int N = 1 << 20;  // Set N = 2^20 = 1,048,576 elements
  float *a, *b, *c; 

  // Allocate unified memory accessible by both CPU and GPU
  hipMallocManaged(&a, N * sizeof(float));
  hipMallocManaged(&b, N * sizeof(float));
  hipMallocManaged(&c, N * sizeof(float));

  // Initialize vectors a and b
  for (int i = 0; i < N; ++i) { a[i] = i; b[i] = i; }

  // Launch the kernel with enough blocks and threads to cover N elements
  // 256 threads per block; (N + 255)/256 ensures we round up
  vecAdd<<<(N + 255)/256, 256>>>(a, b, c, N);

  // Wait for the GPU to finish before accessing results
  hipDeviceSynchronize();

  // Output the value of c[42] to verify correctness (should be 42 + 42 = 84)
  std::cout << "c[42] = " << c[42] << std::endl;

  hipFree(a); hipFree(b); hipFree(c);
  return 0;
}
