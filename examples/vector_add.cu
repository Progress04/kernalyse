
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vecAdd(float *a, float *b, float *c, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) c[i] = a[i] + b[i];
}

int main() {
  int N = 1 << 20;
  float *a, *b, *c;
  hipMallocManaged(&a, N * sizeof(float));
  hipMallocManaged(&b, N * sizeof(float));
  hipMallocManaged(&c, N * sizeof(float));

  for (int i = 0; i < N; ++i) { a[i] = i; b[i] = i; }

  vecAdd<<<(N + 255)/256, 256>>>(a, b, c, N);
  hipDeviceSynchronize();

  std::cout << "c[42] = " << c[42] << std::endl;

  hipFree(a); hipFree(b); hipFree(c);
  return 0;
}
