
#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel for vector addition
__global__ void vecAdd(float *a, float *b, float *c, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) c[i] = a[i] + b[i];
}

// CUDA kernel for vector multiplication
__global__ void vecMul(float *a, float *b, float *c, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) c[i] = a[i] * b[i];
}

// CUDA kernel for vector squaring
__global__ void vecSquare(float *a, float *b, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) b[i] = a[i] * a[i];
}

int main() {
  int N = 1 << 20;
  float *a, *b, *c;

  hipMallocManaged(&a, N * sizeof(float));
  hipMallocManaged(&b, N * sizeof(float));
  hipMallocManaged(&c, N * sizeof(float));

  for (int i = 0; i < N; ++i) {
    a[i] = i * 0.5f;
    b[i] = i * 0.25f;
  }

  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

  vecAdd<<<blocksPerGrid, threadsPerBlock>>>(a, b, c, N);
  hipDeviceSynchronize();

  vecMul<<<blocksPerGrid, threadsPerBlock>>>(a, b, c, N);
  hipDeviceSynchronize();

  vecSquare<<<blocksPerGrid, threadsPerBlock>>>(a, b, N);
  hipDeviceSynchronize();

  std::cout << "c[42] after vecMul = " << c[42] << std::endl;
  std::cout << "b[42] after vecSquare = " << b[42] << std::endl;

  hipFree(a);
  hipFree(b);
  hipFree(c);

  return 0;
}
